#include "../runtime_api.hpp"

#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include "utils.cuh"

namespace llaisys::device::nvidia {

namespace runtime_api {
int getDeviceCount() {
    int device_count = 0;
    CHECK_CUDA(hipGetDeviceCount(&device_count));
    return device_count;
}

void setDevice(int device_id) {
    CHECK_CUDA(hipSetDevice(device_id));
}

void deviceSynchronize() {
    CHECK_CUDA(hipDeviceSynchronize());
}

llaisysStream_t createStream() {
    hipStream_t hip_stream;
    CHECK_CUDA(hipStreamCreate(&hip_stream));
    return static_cast<llaisysStream_t>(hip_stream);
}

void destroyStream(llaisysStream_t stream) {
    hipStream_t hip_stream = static_cast<hipStream_t>(stream);
    CHECK_CUDA(hipStreamDestroy(hip_stream)); 
}
void streamSynchronize(llaisysStream_t stream) {
    CHECK_CUDA(hipStreamSynchronize(static_cast<hipStream_t>(stream)));
}

void *mallocDevice(size_t size) {
    void *ptr = nullptr;
    CHECK_CUDA(hipMalloc(&ptr, size));
    return ptr;
}

void freeDevice(void *ptr) {
    if(ptr) {
        CHECK_CUDA(hipFree(ptr));
    }
}

// 分配锁页内存
void* mallocHost(size_t size) { 
    void* ptr = nullptr;
        CHECK_CUDA(hipHostMalloc(&ptr, size));
    return ptr;
}


void freeHost(void *ptr) {
    if(ptr) {       
        CHECK_CUDA(hipHostFree(ptr));
    }
}

void memcpySync(void *dst, const void *src, size_t size, llaisysMemcpyKind_t kind) {
    CHECK_CUDA(hipMemcpy(dst,
                          src,
                          size,
                          static_cast<hipMemcpyKind>(kind)));
}

void memcpyAsync(void *dst, const void *src, size_t size, llaisysMemcpyKind_t kind, llaisysStream_t stream) {
    CHECK_CUDA(hipMemcpyAsync(dst,
                               src,
                               size,
                               static_cast<hipMemcpyKind>(kind),
                               static_cast<hipStream_t>(stream)));
}

static const LlaisysRuntimeAPI RUNTIME_API = {
    &getDeviceCount,
    &setDevice,
    &deviceSynchronize,
    &createStream,
    &destroyStream,
    &streamSynchronize,
    &mallocDevice,
    &freeDevice,
    &mallocHost,
    &freeHost,
    &memcpySync,
    &memcpyAsync};

} // namespace runtime_api

const LlaisysRuntimeAPI *getRuntimeAPI() {
    return &runtime_api::RUNTIME_API;
}
} // namespace llaisys::device::nvidia
