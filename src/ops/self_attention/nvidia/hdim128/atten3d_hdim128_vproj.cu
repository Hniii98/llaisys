#include "hip/hip_runtime.h"
#include "llaisys.h"

#include "../self_attention_kernels.cuh"
#include "../../../../device/nvidia/utils.cuh"
#include "../../../../core/llaisys_core.hpp"



#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hipblaslt.h>

#include <vector>


namespace llaisys::ops::nvidia::kernels {

template<typename T, hipDataType CType>
void atten3d_hdim128_vproj_kernel(
    T*       atten_val,   // [seqlen, nhead, d=128]  row-major
    const T* v,           // [total_len, nkvhead, d=128] row-major
    const float* score,   // [seqlen, nhead, total_len]  row-major
    size_t   seq_len,
    size_t   nhead,
    size_t   total_len,
    size_t   nkvhead,
    hipStream_t stream_in) {

    constexpr int HDIM = 128;

	ASSERT(seq_len && nhead && total_len && nkvhead, "bad dims");
    ASSERT((nhead % nkvhead) == 0, "GQA requires nhead % nkvhead == 0");
    ASSERT(atten_val && v && score, "null device ptr");

    // cuBLASLt 句柄
    hipblasLtHandle_t ltHandle;
    CHECK_CUBLAS(hipblasLtCreate(&ltHandle));

	

    // Matmul 描述符
    hipblasLtMatmulDesc_t opDesc;
    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    {
        hipblasOperation_t transa = HIPBLAS_OP_N; // A = score: [M=seqlen, K=total_len]
        hipblasOperation_t transb = HIPBLAS_OP_N; // B = V    : [K=total_len, N=HDIM]
        CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(
            opDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
        CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(
            opDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));
  
    }


    // 矩阵布局 row-major
    hipblasLtOrder_t order = HIPBLASLT_ORDER_ROW;

    // A: [M=seqlen, K=total_len], dtype=FP32
	hipblasLtMatrixLayout_t Adesc;
	CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(
		&Adesc, HIP_R_32F,
		/* rows = M */ static_cast<int64_t>(seq_len),
		/* cols = K */ static_cast<int64_t>(total_len),
		/* ld   = K */ static_cast<int64_t>(nhead * total_len)));
	CHECK_CUBLAS(hipblasLtMatrixLayoutSetAttribute(
		Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));

	// B: [K=total_len, N=HDIM], dtype=CType
	hipblasLtMatrixLayout_t Bdesc;
	CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(
		&Bdesc, CType,
		/* rows = K */ static_cast<int64_t>(total_len),
		/* cols = N */ static_cast<int64_t>(HDIM),
		/* ld   = N */ static_cast<int64_t>(nkvhead * HDIM)));
	CHECK_CUBLAS(hipblasLtMatrixLayoutSetAttribute(
		Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));

	// D: [M=seqlen, N=HDIM], dtype=CType
	hipblasLtMatrixLayout_t Ddesc;
	CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(
		&Ddesc, CType,
		/* rows = M */ static_cast<int64_t>(seq_len),
		/* cols = N */ static_cast<int64_t>(HDIM),
		/* ld   = N */ static_cast<int64_t>(nhead * HDIM)));
	CHECK_CUBLAS(hipblasLtMatrixLayoutSetAttribute(
		Ddesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));


    // 算法选择
    hipblasLtMatmulPreference_t preference;
    CHECK_CUBLAS(hipblasLtMatmulPreferenceCreate(&preference));
    size_t workspaceSize = 1 << 22; // 4MB
    CHECK_CUBLAS(hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
        &workspaceSize, sizeof(workspaceSize)));

    // RAII workspace
    auto workspace_storage =
        llaisys::core::context().runtime().allocateDeviceStorage(workspaceSize);
    void* workspace = static_cast<void*>(workspace_storage->memory()); // 允许为空，库会降级算法

	ASSERT(workspace_storage, "workspace empty!");

    hipblasLtMatmulHeuristicResult_t heuristic{};
    int returnedResults = 0;
    CHECK_CUBLAS(hipblasLtMatmulAlgoGetHeuristic(
        ltHandle, opDesc,
        Adesc, Bdesc, /*Cdesc*/ nullptr, Ddesc,
        preference, 1, &heuristic, &returnedResults));
    if (returnedResults == 0) {
        hipblasLtMatmulPreferenceDestroy(preference);
        hipblasLtMatrixLayoutDestroy(Adesc);
        hipblasLtMatrixLayoutDestroy(Bdesc);
        hipblasLtMatrixLayoutDestroy(Ddesc);
        hipblasLtDestroy(ltHandle);
        throw std::runtime_error("hipblasLtMatmul: no heuristic result");
    }

    // 5) loop over heads (GQA: nkvhead < nhead)
    const float alpha = 1.0f;
    const float beta  = 0.0f;

    // nhead nkvhead, qwen2中分别为12和2能够整除
    

    for (size_t h = 0; h < nhead; ++h) {
        const size_t kvh = (h * nkvhead) / nhead; // head -> kv_head

        // row-major 
        const float* A = score     + h * total_len;
        const T*     B = v         + kvh * HDIM;
        T*           D = atten_val + h * HDIM;

        CHECK_CUBLAS(hipblasLtMatmul(
            ltHandle,
            opDesc,
            &alpha,
            A, Adesc,        // const float* -> const void*
            B, Bdesc,        // const T* -> const void* 
            &beta,
            D, Ddesc, 
            D, Ddesc,        // T* -> void*     
            &heuristic.algo,
            workspace, workspaceSize,
            stream_in));
    }
	CHECK_CUDA(hipGetLastError());
	CHECK_CUDA(hipStreamSynchronize(stream_in));


	hipblasLtMatmulPreferenceDestroy(preference);
    hipblasLtMatrixLayoutDestroy(Adesc);
    hipblasLtMatrixLayoutDestroy(Bdesc);
    hipblasLtMatrixLayoutDestroy(Ddesc);
	hipblasLtMatmulDescDestroy(opDesc);
    hipblasLtDestroy(ltHandle);

}


// 显式实例化
template void atten3d_hdim128_vproj_kernel<float, HIP_R_32F>(
    float*, const float*, const float*, size_t, size_t, size_t, size_t, hipStream_t);

template void atten3d_hdim128_vproj_kernel<__half, HIP_R_16F>(
    __half*, const __half*, const float*, size_t, size_t, size_t, size_t, hipStream_t);

template void atten3d_hdim128_vproj_kernel<__hip_bfloat16, HIP_R_16BF>(
    __hip_bfloat16*, const __hip_bfloat16*, const float*, size_t, size_t, size_t, size_t, hipStream_t);



} // namespace llaisys::ops::nvidia::kernels
